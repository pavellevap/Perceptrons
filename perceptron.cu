#include "hip/hip_runtime.h"
#include "perceptron.h"


/**
 *=======================================================================================================
 *                                  Элементарный перцептрон
 *=======================================================================================================
 */


void SaveElementaryPerceptron(const ElementaryPerceptronData& pd, string FileName) {
    IO io;
    io.openOF(FileName.c_str());
    io.writet(pd.amountOfS, sizeof(pd.amountOfS) * 8);
    io.writet(pd.amountOfA, sizeof(pd.amountOfA) * 8);
    io.writet(pd.amountOfR, sizeof(pd.amountOfR) * 8);
    for (size_t i = 0; i < pd.amountOfA; i++)
        for (size_t j = 0; j < pd.amountOfS; j++)
            if (pd.ASEdges[i][j] == 1)
                io.writebit(1);
            else
                io.writebit(0);
    for (size_t i = 0; i < pd.amountOfR; i++)
        for (size_t j = 0; j < pd.amountOfA; j++)
            io.writet(pd.RAEdges[i][j], sizeof(pd.RAEdges[i][j]) * 8);
    io.closeOF();
}

void LoadElementaryPerceptron(ElementaryPerceptronData& pd, string FileName) {
    IO io;
    io.openIF(FileName.c_str());
    io.readt(pd.amountOfS, sizeof(pd.amountOfS) * 8);
    io.readt(pd.amountOfA, sizeof(pd.amountOfA) * 8);
    io.readt(pd.amountOfR, sizeof(pd.amountOfR) * 8);

    pd.ASEdges = new short*[pd.amountOfA];
    for (size_t i = 0; i < pd.amountOfA; i++) {
        pd.ASEdges[i] = new short[pd.amountOfS];
        for (size_t j = 0; j < pd.amountOfS; j++) {
            uchar bit;
            io.readbit(bit);
            if (bit)
                pd.ASEdges[i][j] = 1;
            else
                pd.ASEdges[i][j] = -1;
        }
    }
    pd.RAEdges = new short*[pd.amountOfR];
    for (size_t i = 0; i < pd.amountOfR; i++) {
        pd.RAEdges[i] = new short[pd.amountOfA];
        for (size_t j = 0; j < pd.amountOfA; j++)
            io.readt(pd.RAEdges[i][j], sizeof(pd.RAEdges[i][j]) * 8);
    }
    io.closeIF();
}

ElementaryPerceptronData::~ElementaryPerceptronData() {
    if (ASEdges) {
        for (size_t i = 0; i < amountOfA; i++)
            if (ASEdges[i])
                delete ASEdges[i];
        delete ASEdges;
    }

    if (RAEdges) {
        for (size_t i = 0; i < amountOfR; i++)
            if (RAEdges[i])
                delete RAEdges[i];
        delete RAEdges;
    }
}

cudaElementaryPerceptron::cudaElementaryPerceptron() { }

cudaElementaryPerceptron::cudaElementaryPerceptron(size_t amountOfS, size_t amountOfA, size_t amountOfR) {
	initialize(amountOfS, amountOfA, amountOfR);
}

__global__ void generateASLayerKernel(short** dev_ASEdges, size_t amountOfS, int seed) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int index = y + amountOfS * x;

	int randomNumber = seed + index;
	randomNumber = (randomNumber * randomNumber + randomNumber) % 1000000007;
	randomNumber = (randomNumber + 1000000007) % 1000000007;
	randomNumber = ((randomNumber * 214013L + 2531011L) >> 16) & 0x7fff;

	dev_ASEdges[x][y] = (randomNumber & 1) ? 1 : -1;
}

void cudaElementaryPerceptron::initialize(size_t amountOfS, size_t amountOfA, size_t amountOfR){
	this->amountOfS = amountOfS;
	this->amountOfA = amountOfA;
    this->amountOfR = amountOfR;

    hipMalloc((void**)&dev_AOutput, amountOfA * sizeof(bool));
    ROutput = new int[amountOfR];
    input = new bool[amountOfS];

    short** ptr;

    hipMalloc((void**)&dev_ASEdges, amountOfA * sizeof(short*));
    ptr = new short*[amountOfA];
    for (size_t i = 0; i < amountOfA; i++)
    	hipMalloc((void**)&ptr[i], amountOfS * sizeof(short));
    hipMemcpy(dev_ASEdges, ptr, amountOfA * sizeof(short*), hipMemcpyHostToDevice);
    delete ptr;

    generateASLayerKernel<<<dim3(amountOfA, amountOfS), 1>>>(dev_ASEdges, amountOfS, clock());

    hipMalloc((void**)&dev_RAEdges, amountOfR * sizeof(short*));
    ptr = new short*[amountOfR];
    for (size_t i = 0; i < amountOfR; i++) {
        hipMalloc((void**)&ptr[i], amountOfA * sizeof(short));
        hipMemset(ptr[i], 0, amountOfA * sizeof(short));
    }
    hipMemcpy(dev_RAEdges, ptr, amountOfR * sizeof(short*), hipMemcpyHostToDevice);
    delete ptr;
}

cudaElementaryPerceptron::~cudaElementaryPerceptron() {
	hipFree(dev_AOutput);
	delete ROutput;
	delete input;

	short** ptr;

	if (dev_ASEdges) {
		ptr = new short*[amountOfA];
		hipMemcpy(ptr, dev_ASEdges, amountOfA * sizeof(short*), hipMemcpyDeviceToHost);
		for (size_t i = 0; i < amountOfA; i++)
			if (ptr[i])
				hipFree(ptr[i]);
		delete ptr;
	    hipFree(dev_ASEdges);
	}

	if (dev_RAEdges) {
		ptr = new short*[amountOfR];
		hipMemcpy(ptr, dev_RAEdges, amountOfR * sizeof(short*), hipMemcpyDeviceToHost);
		for (size_t i = 0; i < amountOfR; i++)
			if (ptr[i])
				hipFree(ptr[i]);
		delete ptr;
		hipFree(dev_RAEdges);
	}
}

void cudaElementaryPerceptron::restoreElementaryPerceptron(const ElementaryPerceptronData& pd) {
	this->~cudaElementaryPerceptron();

    amountOfR = pd.amountOfR;
    amountOfA = pd.amountOfA;
    amountOfS = pd.amountOfS;

    hipMalloc((void**)&dev_AOutput, amountOfA * sizeof(bool));
    ROutput = new int[amountOfR];
    input = new bool[amountOfS];

    short** ptr;

    hipMalloc((void**)&dev_ASEdges, amountOfA * sizeof(short*));
    ptr = new short*[amountOfA];
    for (size_t i = 0; i < amountOfA; i++) {
        hipMalloc((void**)&ptr[i], amountOfS * sizeof(short));
        hipMemcpy(ptr[i], pd.ASEdges[i], sizeof(short) * amountOfS, hipMemcpyHostToDevice);
    }
    hipMemcpy(dev_ASEdges, ptr, amountOfA * sizeof(short*), hipMemcpyHostToDevice);
    delete ptr;

    hipMalloc((void**)&dev_RAEdges, amountOfR * sizeof(short*));
    ptr = new short*[amountOfR];
    for (size_t i = 0; i < amountOfR; i++) {
        hipMalloc((void**)&ptr[i], amountOfA * sizeof(short));
        hipMemcpy(ptr[i], pd.RAEdges[i], sizeof(short) * amountOfA, hipMemcpyHostToDevice);
    }
    hipMemcpy(dev_RAEdges, ptr, amountOfR * sizeof(short*), hipMemcpyHostToDevice);
    delete ptr;
}

ElementaryPerceptronData cudaElementaryPerceptron::getElementaryPerceptronData() {
    ElementaryPerceptronData pd;
    pd.amountOfR = amountOfR;
    pd.amountOfA = amountOfA;
    pd.amountOfS = amountOfS;

    short** ptr;

    ptr = new short*[amountOfA];
    hipMemcpy(ptr, dev_ASEdges, amountOfA * sizeof(short*), hipMemcpyDeviceToHost);
    pd.ASEdges = new short*[amountOfA];
    for (size_t i = 0; i < amountOfA; i++) {
        pd.ASEdges[i] = new short[amountOfS];
        hipMemcpy(pd.ASEdges[i], ptr[i], sizeof(short) * amountOfS, hipMemcpyDeviceToHost);
    }
    delete ptr;

    ptr = new short*[amountOfR];
    hipMemcpy(ptr, dev_RAEdges, amountOfR * sizeof(short*), hipMemcpyDeviceToHost);
    pd.RAEdges = new short*[amountOfR];
    for (size_t i = 0; i < amountOfR; i++) {
        pd.RAEdges[i] = new short[amountOfA];
        hipMemcpy(pd.RAEdges[i], ptr[i], sizeof(short) * amountOfA, hipMemcpyDeviceToHost);
    }
    delete ptr;

    return pd;
}

void cudaElementaryPerceptron::setInput(bool* in) {
    memcpy(input, in, amountOfS * sizeof(bool));
}

void cudaElementaryPerceptron::setInput(size_t index, bool value) {
    if (index >= amountOfS)
        cerr << "Выход за границу массива в функции cudaElementaryPerceptron::setInput()\n";
    input[index] = value;
}

void cudaElementaryPerceptron::setAOutput(bool* out) {
    hipMemcpy(dev_AOutput, out, amountOfA * sizeof(bool), hipMemcpyHostToDevice);
}

void cudaElementaryPerceptron::setAOutput(size_t index, bool value) {
    if (index >= amountOfA)
        cerr << "Выход за границу массива в функции cudaElementaryPerceptron::setAOutput()\n";
    hipMemcpy(dev_AOutput + index, &value, sizeof(value), hipMemcpyHostToDevice);
}

void __global__ calculateAOutputKernel(bool* dev_input, bool* dev_AOutput, size_t amountOfS, size_t amountOfA, short** dev_ASEdges) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < amountOfA) {
		int sum = 0;
		for (int i = 0; i < amountOfS; i++)
			if (dev_input[i])
				sum += dev_ASEdges[index][i];
		dev_AOutput[index] = sum > 0;
	}
}



void cudaElementaryPerceptron::calculateAOutput() {
	bool* dev_input;
	hipMalloc((void**)&dev_input, amountOfS * sizeof(bool));
	hipMemcpy(dev_input, input, amountOfS * sizeof(bool), hipMemcpyHostToDevice);

	int amountOfBlocks = (amountOfA + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    calculateAOutputKernel<<<amountOfBlocks, THREADS_PER_BLOCK>>>(dev_input, dev_AOutput, amountOfS, amountOfA, dev_ASEdges);
}

__global__ void calculateROutputKernel(int RIndex, bool* dev_AOutput, int* dev_sum, size_t dev_amountOfA, short** dev_RAEdges) {
	__shared__ int tmp[THREADS_PER_BLOCK];

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < dev_amountOfA && dev_AOutput[index])
		tmp[threadIdx.x] = dev_RAEdges[RIndex][index];
	else
		tmp[threadIdx.x] = 0;


	__syncthreads();

	int i = THREADS_PER_BLOCK >> 1;
	while (i) {
		if (threadIdx.x < i)
			tmp[threadIdx.x] += tmp[threadIdx.x + i];
		i >>= 1;
		__syncthreads();
	}
	if (threadIdx.x == 0)
		dev_sum[blockIdx.x] = tmp[0];
}

void cudaElementaryPerceptron::calculateROutput() {
	int amountOfBlocks = (amountOfA + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	int* sum = new int[amountOfBlocks];
	int* dev_sum;
	hipMalloc((void**)&dev_sum, amountOfBlocks * sizeof(int));

    for (size_t i = 0; i < amountOfR; ++i) {
        calculateROutputKernel<<<amountOfBlocks, THREADS_PER_BLOCK>>>(i, dev_AOutput, dev_sum, amountOfA, dev_RAEdges);

        hipMemcpy(sum, dev_sum, amountOfBlocks * sizeof(int), hipMemcpyDeviceToHost);

        ROutput[i] = 0;
        for (size_t j = 0; j < amountOfBlocks; j++)
        	ROutput[i] += sum[j];
    }

    delete sum;
    hipFree(dev_sum);
}

void cudaElementaryPerceptron::calculateOutput() {
	/*float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);*/
    calculateAOutput();
    /*hipEventRecord(stop, 0);
    hipEventSynchronize(stop);*/

    /*hipEventElapsedTime(&elapsedTime, start, stop);
    printf("%.6f	", elapsedTime);

    hipEventRecord(start, 0);*/
    calculateROutput();
    /*hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("%.6f	", elapsedTime);*/

}

__global__ void correctKernel(int RIndex, int add, bool* dev_AOutput, short** dev_RAEdges) {
	int index = blockIdx.x;
	if (dev_AOutput[index])
		dev_RAEdges[RIndex][index] += add;
}

void cudaElementaryPerceptron::correct(size_t index, int add) {
    if (index >= amountOfR)
        cerr << "Выход за границы массива в функции cudaElementaryPerceptron::correct()\n";

    correctKernel<<<amountOfA, 1>>>(index, add, dev_AOutput, dev_RAEdges);
}

void cudaElementaryPerceptron::teach(int* desierdOutput) {
	//#pragma omp parallel for                 /// ?????
    for (size_t i = 0; i < amountOfR; i++) {
        bool ans1 = ROutput[i] > 0;
        bool ans2 = desierdOutput[i] > 0;
        if (ans1 != ans2)
            if (ROutput[i] > 0)
                correct(i, -1);
            else
                correct(i, 1);
    }
}

bool* cudaElementaryPerceptron::getAOutput() {
    bool* output = new bool[amountOfA];
    hipMemcpy(output, dev_AOutput, sizeof(bool) * amountOfA, hipMemcpyDeviceToHost);
    return output;
}

bool cudaElementaryPerceptron::getAOutput(size_t index) {
    if (index >= amountOfA)
        cerr << "Выход за границу массива в функции cudaElementaryPerceptron::getAOutput()\n";
    bool value;
    hipMemcpy(&value, dev_AOutput + index, sizeof(value), hipMemcpyDeviceToHost);

    return value;
}

int* cudaElementaryPerceptron::getROutput() {
    int* output = new int[amountOfR];
    memcpy(output, ROutput, sizeof(int) * amountOfR);
    return output;
}

int cudaElementaryPerceptron::getROutput(size_t index) {
    if (index >= amountOfR)
        cerr << "Выход за границу массива в функции cudaElementaryPerceptron::getROutput()\n";
    return ROutput[index];
}

size_t cudaElementaryPerceptron::getAmountOfR() {
    return amountOfR;
}

size_t cudaElementaryPerceptron::getAmountOfA() {
    return amountOfA;
}

size_t cudaElementaryPerceptron::getAmountOfS() {
    return amountOfS;
}

